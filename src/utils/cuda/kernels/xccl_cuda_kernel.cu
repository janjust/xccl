#include "hip/hip_runtime.h"
#include <api/xccl.h>
#include <hip/hip_runtime.h>

__global__ void dummy_kernel(volatile int *stop) {
    int should_stop;
    do {
        should_stop = *stop;
    } while(!should_stop);
    return;
}

#ifdef __cplusplus
extern "C" {
#endif

hipError_t xccl_cuda_dummy_kernel(int *stop, hipStream_t stream)
{
    dummy_kernel<<<1, 1, 0, stream>>>(stop);
    return hipGetLastError();
}

#ifdef __cplusplus
}
#endif
